#include "hip/hip_runtime.h"
#define WITH_CUDA

#include <iostream>
#include <fstream>
#include <random>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "vector/include/Vector.hpp"
#include "cudastreams/CudaEvent.h"
#include "cudastreams/CudaStream.h"
#include "util.hpp"
#include "warp_reduction.hpp"

using index_type = int;
using host_index   = memory::HostVector<index_type>;
using device_index = memory::DeviceVector<index_type>;

using value_type = double;
using host_vector   = memory::HostVector<value_type>;
using device_vector = memory::DeviceVector<value_type>;

bool test(host_vector const& reference, host_vector const& v) {
    assert(reference.size() == v.size());
    auto success = true;
    for(auto i=0; i<reference.size(); ++i) {
        if(reference[i] != v[i]) {
            printf("  error %10d expected %5.1f got %5.1f\n",
                   (int)i, (float)(reference[i]), (float)(v[i]));
            success = false;
        }
    }

    return success;
}

void print(host_vector const& v) {
    auto pos = 0;
    while(pos<v.size()) {
        auto col = 0;
        while(col<32 && pos<v.size()) {
            printf("%3.0f", v[pos]);
            ++pos;
            ++col;
        }
        printf("\n");
    }
}

void print(host_index const& v) {
    auto pos = 0;
    while(pos<v.size()) {
        auto col = 0;
        while(col<32 && pos<v.size()) {
            printf("%3d", v[pos]);
            ++pos;
            ++col;
        }
        printf("\n");
    }
}

host_index generate_index(int n, int max_bucket_size) {
    std::random_device rd;
    std::default_random_engine e(rd());
    std::uniform_int_distribution<int> rng(1, max_bucket_size);

    std::cout << " == bucket size " << max_bucket_size << " ==" << std::endl;
    std::cout << " == array size " << n << " ==" << std::endl;

    // generate the index vector on the host
    host_index index(n);

    auto pos = 0;
    auto m = 0;
    while(pos<n) {
        auto increment = rng(e);
        auto final = std::min(pos+increment, n);
        while(pos<final) {
            index[pos++] = m;
        }
        ++m;
    }

    return index;
}

host_index read_index(std::string fname) {
    std::ifstream fid(fname);
    if(!fid.is_open()) {
        std::cerr << memory::util::red("error") << " : unable to open file "
                  << memory::util::yellow(fname) << std::endl;
        exit(1);
    }

    int n;
    fid >> n;
    std::cout << "loading index of length " << n << " from file " << fname << std::endl;
    host_index index(n);
    for(auto i=0; i<n; ++i) fid >> index[i];
    return index;
}

int main(int argc, char** argv) {
    int max_bucket_size = read_arg(argc, argv, 1, -1);

    // input  array of length n
    // output array of length m
    // sorted indexes in p (length n)
    auto ph =
        max_bucket_size < 1 ?
            read_index("index.txt")
          : generate_index(1<<25, max_bucket_size);
    const auto n = ph.size();
    auto m = ph[n-1];

    // make reference solution
    host_vector solution(m);
    solution(0,m) = 0;
    for(auto i : ph) {
        solution[i] += 1;
    }

    if(n<=256) {
        std::cout << "in \n"; print(ph);
        std::cout << "out\n"; print(solution);
    }

    // configure cuda stream for timing
    CudaStream stream_compute(false);

    // push index to the device
    device_index p = ph;

    device_vector in(n);
    std::vector<device_vector> out(3);
    for(auto &o: out) {
        o = device_vector(m);
    }

    in(memory::all) = value_type{1};

    auto threads_per_block=256;
    auto blocks=(n+threads_per_block-1)/threads_per_block;

    for(auto &o: out) o(memory::all) = value_type{0};
    auto b1 = stream_compute.insert_event();
    gpu::reduce_by_index
        <<<blocks, threads_per_block>>>
        (in.data(), out[0].data(), p.data(), n);
    auto e1 = stream_compute.insert_event();
    e1.wait();
    std::cout << "  1  " << e1.time_since(b1) << " seconds" << std::endl;
    test(solution, host_vector(out[0]));

    for(auto &o: out) o(memory::all) = value_type{0};
    auto b2 = stream_compute.insert_event();
    gpu::reduce_by_index<value_type>
        <<<blocks, threads_per_block>>>
        (in.data(), out[0].data(), out[1].data(), p.data(), n);
    auto e2 = stream_compute.insert_event();
    e2.wait();
    std::cout << "  2  " << e2.time_since(b2) << " seconds" << std::endl;
    test(solution, host_vector(out[0]));
    test(solution, host_vector(out[1]));

    for(auto &o: out) o(memory::all) = value_type{0};
    auto b3 = stream_compute.insert_event();
    gpu::reduce_by_index<value_type>
        <<<blocks, threads_per_block>>>
        (in.data(), out[0].data(), out[1].data(), out[2].data(), p.data(), n);
    auto e3 = stream_compute.insert_event();
    e3.wait();
    std::cout << "  3  " << e3.time_since(b3) << " seconds" << std::endl;
    test(solution, host_vector(out[0]));
    test(solution, host_vector(out[1]));
    test(solution, host_vector(out[2]));

    return 0;
}

